#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
// #include <cuda/std/bfloat16>


#define EMBEDDING_DIM 128  // BLOCK_SIZE * FETCH_PER_WI == EMBEDDING_DIM
#define NUM_EMBEDDINGS 2000000
#define BATCH_SIZE 65536
#define BLOCK_SIZE0 32
#define BLOCK_SIZE1 8
#define NUM_ITERS 20
#define MULTIHOT_SIZE 20    // 每个block处理MULTIHOT_SIZE行

void check_cuda_error(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line
                  << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define CHECK_CUDA(call) check_cuda_error((call), __FILE__, __LINE__)

template<typename scalar_t>
__global__ void embedding_block_kernel(
    const scalar_t* __restrict__ weights,
    const int* __restrict__ indices,
    scalar_t* __restrict__ output) {

    int64_t chunksPerBag = EMBEDDING_DIM / (int64_t)blockDim.x; // 64 / 32 = 2   输入的每个bag，对应多少chunk
    int64_t numChunks = BATCH_SIZE * chunksPerBag;              // 65536 * 2
    int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;// 0-1023 * 8 + 0-7 每个block每次8个chunk
    int64_t chunkStride = gridDim.x * blockDim.y;               // 1024 * 8         每个grid每次1024*8个chunk

    for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
        int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
        int64_t bs = chunk / chunksPerBag;
        const float *weightFeat = weights + featureDim;

        int64_t begin = bs*MULTIHOT_SIZE;
        int64_t end = (bs+1)*MULTIHOT_SIZE;

        scalar_t sum = 0;
        for (int64_t emb = begin; emb < end; emb++) {
            int index = indices[emb];
            sum += weightFeat[index*EMBEDDING_DIM];
        }
        output[bs * EMBEDDING_DIM + featureDim] = sum;
    }

}

template<typename scalar_t>
bool check_out(scalar_t* weights, int* h_indices, scalar_t* h_output, int batch_size) {
    for (int i = 0; i < batch_size; i++) {
        for (int j = 0; j < EMBEDDING_DIM; j++) {
            scalar_t refe = 0;
            for (int k = 0; k < MULTIHOT_SIZE; k++) {
                refe += weights[h_indices[k+MULTIHOT_SIZE*i]*EMBEDDING_DIM + j];
            }
            scalar_t test = h_output[i*EMBEDDING_DIM + j];
            if (test - refe > 1e-3 || test - refe < -1e-3) {
                std::cout << i << " " << j << " " << refe << " " << test << std::endl;
                return false;
            }
        }
    }
    return true;
}

void test_bandwidth() {
    using scalar_t = float;
    // if (EMBEDDING_DIM != BLOCK_SIZE*FETCH_PER_WI) {
    //     return;
    // }
    size_t table_size = NUM_EMBEDDINGS * EMBEDDING_DIM * sizeof(scalar_t);
    size_t output_size = BATCH_SIZE * EMBEDDING_DIM * sizeof(scalar_t);

    scalar_t *h_table = new scalar_t[NUM_EMBEDDINGS * EMBEDDING_DIM];
    int *h_indices = new int[BATCH_SIZE*MULTIHOT_SIZE];
    scalar_t *h_output = new scalar_t[BATCH_SIZE * EMBEDDING_DIM];

    // 初始化数据
    for (int i = 0; i < NUM_EMBEDDINGS * EMBEDDING_DIM; ++i) {
        h_table[i] = static_cast<scalar_t>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < BATCH_SIZE * MULTIHOT_SIZE; ++i) {
        // scalar_t tmp = rand() % NUM_EMBEDDINGS; h_indices[i] = tmp > 0 ? tmp : -tmp;
        h_indices[i] = i % NUM_EMBEDDINGS;
    }

    // 分配设备内存
    scalar_t *d_table, *d_output;
    int *d_indices;
    CHECK_CUDA(hipMalloc(&d_table, table_size));
    CHECK_CUDA(hipMalloc(&d_indices, MULTIHOT_SIZE * BATCH_SIZE * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_output, output_size));

    // 数据传输
    CHECK_CUDA(hipMemcpy(d_table, h_table, table_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_indices, h_indices, MULTIHOT_SIZE * BATCH_SIZE * sizeof(int), hipMemcpyHostToDevice));

    dim3 block = dim3(BLOCK_SIZE0, BLOCK_SIZE1);
    // 预热
    for (int i = 0; i < 5; ++i) {
        embedding_block_kernel<<<BATCH_SIZE, block>>>(d_table, d_indices, d_output);
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // 带宽测试
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < NUM_ITERS; ++i) {
        embedding_block_kernel<<<BATCH_SIZE, block>>>(d_table, d_indices, d_output);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float elapsed_ms;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_ms, start, stop));

    // 计算带宽
    float total_bytes = NUM_ITERS * BATCH_SIZE * (EMBEDDING_DIM * sizeof(scalar_t)) * (MULTIHOT_SIZE + 1);
    float bandwidth = (total_bytes / (elapsed_ms / 1000.0f)) / (1024 * 1024 * 1024);

    CHECK_CUDA(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost));

    std::cout << "Total Bytes: " << total_bytes / (1024 * 1024 * 1024) << " GB" << std::endl;
    std::cout << "Global Memory Bandwidth: " << bandwidth << " GB/s" << std::endl;
    std::cout << "Execution Time: " << elapsed_ms / NUM_ITERS << " ms per batch" << std::endl;

    if (!check_out(h_table, h_indices, h_output, BATCH_SIZE)) {
        std::cout << "error\n";
    } else {
        std::cout << "accuracy pass\n";
    }

    // 清理资源
    delete[] h_table;
    delete[] h_indices;
    delete[] h_output;
    CHECK_CUDA(hipFree(d_table));
    CHECK_CUDA(hipFree(d_indices));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
}

int main() {
    test_bandwidth();
    return 0;
}